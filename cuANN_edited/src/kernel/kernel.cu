#include "hip/hip_runtime.h"
#include "kernel.hpp"
namespace cuANN
{

__global__ void forward_prop ( 
                               const float * weight, // W[ji] 
                               const float * input,  // O[j]
                               float * output,       // I[i]
                               unsigned int w_size   // weights per node (# of columns)
                             )
{
    // X is input size (w_size)
    int x = blockIdx.x * blockDim.x + threadIdx.x; 
    // Y is weights per neuron/node (i_size)
    int y = blockIdx.y * blockDim.y + threadIdx.y;   
    //  I[j] * W[i] - Row-Major Matrix
    output[w_size*x+y] = __fmul_rz(input[x], weight[w_size * x + y]);
}

__global__ void sum_columns ( 
                                float * w_mtx,
                                float * output, 
                                unsigned int height,
                                unsigned int width
                            )
{
    // X thread iterates Columns and sums their Row values
    int x = blockIdx.x * blockDim.x + threadIdx.x; 
    float total;
    for ( int y = 0; y < height; y++ )
    {
        total = __fadd_rz( total, w_mtx[y*width+x]);
    }
    output[x] = total;
}

__global__ void delta_output (
                                const float * primed_sum,
                                const float * ideal,
                                const float * actual,
                                float * delta,
                                unsigned int index
                             )
{
    // x is the output neuron/node count (e.g., length of actual & ideal)
    int x = blockIdx.x * blockDim.x + threadIdx.x;

    // Calculate the Negative Error: -(Actual - Ideal)
    float neg_error = __fmul_rz(-1,(actual[x] - ideal[x]));

    // -E * σ'(Σ(O[i])
    delta[x+index] = __fmul_rz( neg_error, primed_sum[x+index] );
}

__global__ void delta_product (
                                const float * w_ik,
                                const float * d_k,
                                float * output,
                                unsigned int width
                              )
{
    // X is layer[i] nodes (size_i)
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    // Y is layer[k] nodes (size_k) == d_k == w_per_n
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    //  W[ik] * δ[k] - Row-Major Matrix
    output[width*x+y] = __fmul_rz( d_k[y], w_ik[width*x+y]);
}

__global__ void delta_sum_rows (
                                float * w_ik_d,
                                float * delta_i,
                                unsigned int width
                               )
{
    // X thread iterates Rows and Sums the respective Column values
    int x = blockIdx.x * blockDim.x + threadIdx.x; 
    float total = 0.f;
    for ( int y = 0; y < width; y++ )
    {
        //printf("X:%d, Σ: %.9f + %.9f\n",x,total,w_ik_d[x*width+y]);
        total = __fadd_rz( total, w_ik_d[x*width+y]);
    }
    delta_i[x] = total;
}

__global__ void delta_hidden (
                               float * prime_ji,
                               float * delta_i
                             )
{
    // X grid is size_i
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    // δ[i] = f'( Σ[ji]) * Σ(w[ik] * δ[k])
    // NOTE: delta_i ALREADY contains `Σ(w[ik] * δ[k])`
    float rhs = delta_i[x];
    // δ[i] = σ'( Σ[ji]) * Σ(w[ik] * δ[k])
    delta_i[x] = __fmul_rz( prime_ji[x], rhs );
}

__global__ void gradient_descent (
                                    float * d_k,
                                    float * o_i,
                                    float * g_ik,
                                    unsigned int size_d
                                 )
{
    // X = Node Delta Count (layer k)
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    // Y = Node Output Count (layer i)
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    // Row-Major Matrix
    g_ik[size_d*x+y] = __fmul_rz( d_k[x], o_i[y]);
}

__global__ void sum_gradients (
                                float * gradient,
                                float * new_value
                              ) 
{
    // X Grid iterates all gradient values 
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    // A Simple summation
    gradient[x] = __fadd_rz( gradient[x], new_value[x] );
}

__global__ void back_prop (
                            float * weight,
                            float * gradient,
                            float * update,
                            float alpha,
                            float epsilon
                         )
{
    // X Grid iterates weight, gradient and update (all same size)    
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    // ε * ( ∂E / ∂W[ik] )
    float lhs = __fmul_rz( epsilon, gradient[x] ); 
    // α * ( Δw(t-1) )
    float rhs = __fmul_rz( alpha, update[x] );
    // Δw(t) = ε * ( ∂E / ∂W[i] ) + α * ( Δw(t-1) )
    float d_w = __fadd_rz( lhs, rhs );
    
    //printf("Δw(t): %f W[i]: %f W[i]+Δw(t): %f Δw(t-1): %f\n",d_w,weight[x],__fadd_rz(weight[x],d_w),update[x]);

    // Update weight: W[i] = W[i] + Δw(t)
    weight[x] = __fadd_rz( weight[x], d_w );
    // Set `Δw(t-1) = Δw(t)`
    update[x] = d_w;
}

__global__ void squared_error ( 
                                const float * ideal,
                                float * actual, 
                                float * errors
                            )
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    float diff = ideal[x] - actual[x];
    errors[x] = __fmul_rz(diff,diff);
    //printf("squared_error: %f, ideal: %f, actual: %f\n",errors[x],ideal[x],actual[x]);
}

};
